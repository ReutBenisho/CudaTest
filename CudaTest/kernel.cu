﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <thread>
#include <vector>
#include <time.h>
#include <iostream>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void addArraysSingleThreadCPU(const int* a, const int* b, int* c, int size, double& time) 
{
    clock_t t = clock();
    for (int i = 0; i < size; ++i)
        c[i] = a[i] + b[i];
    t = clock() - t;
    time = (((double)t) / CLOCKS_PER_SEC) ;  // in seconds
}

void addArraysMultiThreadCPU(const int* a, const int* b, int* c, int i)
{
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySizeToPrint = 5;
    
    int arraySize = 0;
    std::cout << "Enter array size:\n" << std::endl;
    std::cin >> arraySize;
    while (arraySize > 0)
    {
        int* a = new int[arraySize];
        for (int i = 0; i < arraySize; ++i)
            a[i] = i + 1;
        int* b = new int[arraySize];
        for (int i = 0; i < arraySize; ++i)
            b[i] = (i + 1) * 10;
        int* cCUDA = new int[arraySize];
        int* cSingleThreadCPU = new int[arraySize];
        int* cMultiThreadCPU = new int[arraySize];
        for (int i = 0; i < arraySize; ++i)
            cCUDA[i] = cSingleThreadCPU[i] = cMultiThreadCPU[i] = 0;
        std::vector<std::thread> threads(arraySize);
        clock_t t;
        double time_taken = 0;

        // Add vectors in parallel.
        t = clock();
        hipError_t cudaStatus = addWithCuda(cCUDA, a, b, arraySize);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addWithCuda failed!\n");
            //return 1;
        }
        t = clock() - t;
        time_taken = (((double)t) / CLOCKS_PER_SEC) ; // in seconds

        printf("\nCUDA took %f seconds: ", time_taken);
        for (int i = 0; i < arraySizeToPrint; ++i)
            printf("%d ", cCUDA[i]);

        //Add vectors in CPU Single thread
        addArraysSingleThreadCPU(a, b, cSingleThreadCPU, arraySize, time_taken);

        printf("\nCPU Single Thread took %f seconds: ", time_taken);
        for (int i = 0; i < arraySizeToPrint; ++i)
            printf("%d ", cSingleThreadCPU[i]);

        //Add vectors in CPU Multi thread

        //t = clock();
        //for (int i = 0; i < arraySize; ++i)
        //    threads[i] = std::thread(addArraysMultiThreadCPU, a, b, cMultiThreadCPU, i);
        //for (auto& th : threads) {
        //    th.join();
        //}
        //t = clock() - t;
        //time_taken = (((double)t) / CLOCKS_PER_SEC) ;  // in seconds

        //printf("\nCPU Multi Thread took %f seconds: ", time_taken);
        //for (int i = 0; i < arraySizeToPrint; ++i)
        //    printf("%d ", cMultiThreadCPU[i]);

        // cudaDeviceReset must be called before exiting in order for profiling and
        // tracing tools such as Nsight and Visual Profiler to show complete traces.
        cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!\n");
            //return 1;
        }
        delete[] a;
        delete[] b;
        delete[] cCUDA;
        delete[] cSingleThreadCPU;
        delete[] cMultiThreadCPU;

        std::cout << "\n\nEnter array size:" << std::endl;
        std::cin >> arraySize;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
